#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include "utils.h"

__global__
void histogram_kernel(unsigned int* d_bins, const float* d_in, const int bin_count, const float lum_min, const float lum_max, const int size) {  
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    float lum_range = lum_max - lum_min;
    int bin = ((d_in[mid]-lum_min) / lum_range) * bin_count;
    
    atomicAdd(&d_bins[bin], 1);
}

__global__ 
void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    
    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_bins[spot];
          __syncthreads();
          if(spot >= 0)
              d_bins[mid] += val;
          __syncthreads();

    }
}

__global__
void reduce_minmax_kernel(const float* const d_in, float* d_out, const size_t size, int minmax) {
    extern __shared__ float shared[];
    
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x; 
    if(mid < size) {
        shared[tid] = d_in[mid];
    } else {
        if(minmax == 0)
            shared[tid] = FLT_MAX;
        else
            shared[tid] = -FLT_MAX;
    }
    __syncthreads();

    if(mid >= size) {   
        if(tid == 0) {
            if(minmax == 0) 
                d_out[blockIdx.x] = FLT_MAX;
            else
                d_out[blockIdx.x] = -FLT_MAX;
        }
        return;
    }
       
    for(unsigned int s = blockDim.x/2; s > 0; s /= 2) {
        if(tid < s) {
            if(minmax == 0) {
                shared[tid] = min(shared[tid], shared[tid+s]);
            } else {
                shared[tid] = max(shared[tid], shared[tid+s]);
            }
        }
        
        __syncthreads();
    }
    
    if(tid == 0) {
        d_out[blockIdx.x] = shared[0];
    }
}

int get_max_size(int n, int d) {
    return (int)ceil( (float)n/(float)d ) + 1;
}

float reduce_minmax(const float* const d_in, const size_t size, int minmax) {
    int BLOCK_SIZE = 32;
    size_t curr_size = size;
    float* d_curr_in;
    
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice));

    float* d_curr_out;
    dim3 thread_dim(BLOCK_SIZE);
    const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
    
    while(1) {
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(curr_size, BLOCK_SIZE)));
        
        dim3 block_dim(get_max_size(size, BLOCK_SIZE));
        reduce_minmax_kernel<<<block_dim, thread_dim, shared_mem_size>>>(
            d_curr_in,
            d_curr_out,
            curr_size,
            minmax
        );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        if(curr_size <  BLOCK_SIZE) 
            break;
        
        curr_size = get_max_size(curr_size, BLOCK_SIZE);
    }
    
    float h_out;
    hipMemcpy(&h_out, d_curr_out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_curr_out);
    return h_out;
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
    const size_t size = numRows*numCols;
    min_logLum = reduce_minmax(d_logLuminance, size, 0);
    max_logLum = reduce_minmax(d_logLuminance, size, 1);
    
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    printf("numBins %d\n", numBins);
    
    unsigned int* d_bins;
    size_t histo_size = sizeof(unsigned int)*numBins;

    checkCudaErrors(hipMalloc(&d_bins, histo_size));    
    checkCudaErrors(hipMemset(d_bins, 0, histo_size));  
    dim3 thread_dim(1024);
    dim3 hist_block_dim(get_max_size(size, thread_dim.x));
    histogram_kernel<<<hist_block_dim, thread_dim>>>(d_bins, d_logLuminance, numBins, min_logLum, max_logLum, size);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    unsigned int h_out[100];
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
        printf("hist out %d\n", h_out[i]);
    
    dim3 scan_block_dim(get_max_size(numBins, thread_dim.x));

    scan_kernel<<<scan_block_dim, thread_dim>>>(d_bins, numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
        printf("cdf out %d\n", h_out[i]);
    

    hipMemcpy(d_cdf, d_bins, histo_size, hipMemcpyDeviceToDevice);    
    checkCudaErrors(hipFree(d_bins));
}
