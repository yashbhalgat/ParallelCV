#include "hip/hip_runtime.h"
/*************************************************************
* ME766 Project
* Median filter kernels in CUDA
* Team: Yash Bhalgat | Meet Shah
*************************************************************/

#include "median.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <>
const int block_x = 16;
const int block_y = 16;
const int blocksize = 64;
__constant__ int width;
__constant__ int height;
texture<uchar, 1, hipReadModeElementType> texRef;
__device__  void isort(uchar* lhs, int N)
{
	int i, j;
	uchar temp;
	for (i = 1; i < N; ++i)
	{
		j = i - 1;
		temp = lhs[i];
		while (j > -1 && lhs[j] > temp)
		{
			lhs[j + 1] = lhs[j];
			--j;
		}
		lhs[j + 1] = temp;
	}
}
__global__ void cuda_median_fil(uchar* src, uchar* dst)
{
#if 0
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int y = threadID / width;
	int x = threadID % width;
	__shared__ uchar temp[8 * blocksize];
	__shared__ uchar* curr;
	if (threadIdx.x == 0)
	{
		curr = src + y * width;	
	}
	__syncthreads();
	if (threadID < (height - 2) * (width - 2))
	{
		int index = threadIdx.x * 8;
		temp[index + 0] = curr[x - 1];
		temp[index + 1] = curr[x];
		temp[index + 2] = curr[x + 1];
		curr += width;
		temp[index + 3] = curr[x - 1];
		temp[index + 4] = curr[x + 1];
		curr += width;
		temp[index + 5] = curr[x - 1];
		temp[index + 6] = curr[x];
		temp[index + 7] = curr[x + 1];
		isort(temp, 8);
		dst[(y + 1) * width + x + 1] = temp[index + 4];
	}
#else
	int x = threadIdx.x;
	int y = threadIdx.y;
	int index_x = x + blockIdx.x * (blockDim.x - 2);
	int index_y = y + blockIdx.y * blockDim.y + 1;
	if (index_x < width && index_y < height - 1)
	{
		__shared__ uchar temp[block_y * 3][block_x];
		int top, mid, down, change;
		top = tex1Dfetch(texRef, index_x + (index_y - 1) * width);
		mid = tex1Dfetch(texRef, index_x + index_y * width);
		down = tex1Dfetch(texRef, index_x + (index_y + 1) * width);
		if (top < mid)
		{
			change = mid;
			mid = top;
			top = change;
		}
		if (top < down)
		{
			change = down;
			down = top;
			top = change;
		}
		if (mid < down)
		{
			change = down;
			down = mid;
			mid = change;
		}
		int index = 3 * y;
		temp[index][x] = top;
		temp[index + 1][x] = mid;
		temp[index + 2][x] = down;
		__syncthreads();

		if (x > 0 && x < block_x - 1)
		{
			uchar box[3][3];
			for (int i = 0; i < 3; ++i)
			{
				for (int j = -1; j < 2; ++j)
				{
					box[i][j] = temp[index + i][x + j];
				}
			}

			for (int i = 0; i < 3; ++i)
			{
				isort(&box[i][0], 3);
			}


			if (box[0][0] < box[1][1])
			{
				change = box[0][0];
				box[0][0] = box[1][1];
				box[1][1] = change;
			}
			if (box[0][0] < box[2][2])
			{
				change = box[0][0];
				box[0][0] = box[2][2];
				box[2][2] = change;
			}
			if (box[1][1] < box[2][2])
				dst[index_x + index_y * width] = box[2][2];
			else
				dst[index_x + index_y * width] = box[1][1];
		}
	}
#endif
}
void medianGPU(Mat src, Mat& dst)
{
	if (src.type() != CV_8UC1)
	{
		src.convertTo(src, CV_8UC1);
	}
	copyMakeBorder(src, src, 1, 1, 1, 1, BORDER_CONSTANT, 0);
	int size = src.rows * src.cols;
	int count = size * sizeof(uchar);
	uchar* dsrc, *ddst, *hdst;
	hdst = (uchar*)malloc(count);
	memset(hdst, 0, count);
	if (!hdst)
	{
		cerr<<"host memory allocated failed!"<<endl;
		return;
	}
	//allocate device memory
	checkCudaErrors(hipMalloc((void**)&dsrc, count));
	checkCudaErrors(hipMalloc((void**)&ddst, count));
	//copy host to device
	checkCudaErrors(hipMemcpy(dsrc, (uchar*)(src.ptr<uchar>(0)), count, hipMemcpyHostToDevice));
	//width and height
	checkCudaErrors(hipMemcpyToSymbol((const void*)&width, (const void*)&src.cols, sizeof(int), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol((const void*)&height, (const void*)&src.rows, sizeof(int), 0, hipMemcpyHostToDevice));
	//kernel function to do median filter
	dim3 block(blocksize);
	dim3 grid(src.rows * src.cols/block.x + 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	cuda_median_fil<<<grid, block>>>(dsrc, ddst);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	cout<<"elapsed time is "<<elapsed_time<<" ms"<<endl;
	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
	//memcpy from device to host
	checkCudaErrors(hipMemcpy(hdst, ddst, count, hipMemcpyDeviceToHost));
	//copy to dst
	dst = Mat_<uchar>(src.rows, src.cols, hdst).clone();
	//release memory
	hipFree(dsrc);
	hipFree(ddst);
	free(hdst);
}

void medianGPU_opti(Mat src, Mat& dst)
{
	if (src.type() != CV_8UC1)
	{
		src.convertTo(src, CV_8UC1);
	}
	copyMakeBorder(src, src, 1, 1, 1, 1, BORDER_CONSTANT, 0);
	int size = src.rows * src.cols;
	int count = size * sizeof(uchar);
	uchar* dsrc, *ddst, *hdst;
	hdst = (uchar*)malloc(count);
	memset(hdst, 0, count);
	if (!hdst)
	{
		cerr<<"host memory allocated failed!"<<endl;
		return;
	}
	//allocate device memory
	checkCudaErrors(hipMalloc((void**)&dsrc, count));
	checkCudaErrors(hipMalloc((void**)&ddst, count));
	//copy host to device
	checkCudaErrors(hipMemcpy(dsrc, src.data, count, hipMemcpyHostToDevice));
	//bind to texture reference
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
	checkCudaErrors(hipBindTexture(0, &texRef, dsrc, &channelDesc, count));
	//width and height
	checkCudaErrors(hipMemcpyToSymbol((const void*)&width, (const void*)&src.cols, sizeof(int), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol((const void*)&height, (const void*)&src.rows, sizeof(int), 0, hipMemcpyHostToDevice));
	//kernel function to do median filter
	dim3 block(block_x, block_y);
	dim3 grid((src.cols + block_x - 2)/(block_x - 2), (src.rows + block_y - 2)/block_y);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	cuda_median_fil<<<grid, block>>>(dsrc, ddst);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	cout<<"elapsed time is "<<elapsed_time<<" ms"<<endl;
	//unbind memory from texturereference
	hipUnbindTexture(&texRef);
	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
	//memcpy from device to host
	checkCudaErrors(hipMemcpy(hdst, ddst, count, hipMemcpyDeviceToHost));
	//copy to dst
	dst = Mat_<uchar>(src.rows, src.cols, hdst).clone();
	//release memory
	hipFree(dsrc);
	hipFree(ddst);
	free(hdst);
}
void medianCPU(Mat& src, Mat& dst)
{
	if (src.type() != CV_8UC1)
	{
		src.convertTo(src, CV_8UC1);
	}
	dst = Mat::zeros(src.rows, src.cols, CV_8UC1);
	copyMakeBorder(src, src, 1, 1, 1, 1, BORDER_CONSTANT);
	uchar h[256] = {0};
	uchar temp[9];
	int sum, n;
	for (int i = 1; i != src.rows - 1; ++i)
	{
		uchar* pre = src.ptr<uchar>(i - 1);
		uchar* curr = src.ptr<uchar>(i);
		uchar* next = src.ptr<uchar>(i + 1);
		for (int j = 1; j != src.cols - 1; ++j)
		{
			sum = 0, n = 0;
			temp[0] = pre[j - 1];
			temp[1] = pre[j];
			temp[2] = pre[j + 1];
			temp[3] = curr[j - 1];
			temp[4] = curr[j];
			temp[5] = curr[j + 1];
			temp[6] = next[j - 1];
			temp[7] = next[j];
			temp[8] = next[j + 1];
			for (int k = 0; k != 9; ++k)
			{
				h[temp[k]] += 1;
			}
			while (sum < 6)
			{
				sum += h[n];
				++n;
			}
			dst.at<uchar>(i - 1, j - 1) = n - 1;
			for (int k = 0; k != 9; ++k)
			{
				h[temp[k]] &= 0x0;
			}
		}
	}
}
