#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{    
    const int2 p = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                               blockIdx.y * blockDim.y + threadIdx.y);
    const int m = p.y * numCols + p.x; 
    if(p.x >= numCols || p.y >= numRows)
         return;
    float color = 0.0f;
    
    for(int f_y = 0; f_y < filterWidth; f_y++) {
        for(int f_x = 0; f_x < filterWidth; f_x++) {
   
            int c_x = p.x + f_x - filterWidth/2;
            int c_y = p.y + f_y - filterWidth/2;
            c_x = min(max(c_x, 0), numCols - 1);
            c_y = min(max(c_y, 0), numRows - 1);
            float filter_value = filter[f_y*filterWidth + f_x];
            color += filter_value*static_cast<float>(inputChannel[c_y*numCols + c_x]);
        }
    }
    outputChannel[m] = color;
}

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  const int2 p = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                             blockIdx.y * blockDim.y + threadIdx.y);
  const int m = p.y * numCols + p.x;    
  if(p.x >= numCols || p.y >= numRows)
      return;
  redChannel[m]   = inputImageRGBA[m].x;
  greenChannel[m] = inputImageRGBA[m].y;
  blueChannel[m]  = inputImageRGBA[m].z;
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);
  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  checkCudaErrors(hipMalloc(&d_filter, sizeof( float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void gaussian_filter(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  const dim3 blockSize(32, 32);
  const dim3 gridSize(numCols/blockSize.x + 1, numRows/blockSize.y + 1);
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,
                                            numRows,
                                            numCols,
                                            d_red,
                                            d_green,
                                            d_blue);
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(
      d_red,
      d_redBlurred,
      numRows,
      numCols,
      d_filter,
      filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(
      d_blue,
      d_blueBlurred,
      numRows,
      numCols,
      d_filter,
      filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());  
  gaussian_blur<<<gridSize, blockSize>>>(
      d_green,
      d_greenBlurred,
      numRows,
      numCols,
      d_filter,
      filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}